#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Author(s):
// A01705291 Ramses Aguila Flores 
// A01209414 David Flores Diaz
// speedup = 159.37460 ms / 0.00390 ms = 4086528.20 %
// Description: This file contains the code to count the number of
//				even numbers within an array using CUDA.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 100000000
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void sum(int *array, long *result) {
        __shared__ long cache[THREADS];

        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        int cacheIndex = threadIdx.x;

        long acum = 0;
        while (tid < SIZE) {
		if (array[tid] % 2 == 0){ 
                	acum ++;
		}
                tid += blockDim.x * gridDim.x;
        }

        cache[cacheIndex] = acum;

        __syncthreads();

        int lim = blockDim.x / 2;
        while (lim > 0) {
                if (cacheIndex < lim) {
                        cache[cacheIndex] += cache[cacheIndex + lim];
                }
                __syncthreads();
                lim /= 2;
        }

        if (cacheIndex == 0) {
                result[blockIdx.x] = cache[cacheIndex];
        }
}
int main(int argc, char* argv[]) {
        int i, *array, *d_a;
        long *results, *d_r;
        double ms;

        array = (int*) malloc( SIZE * sizeof(int) );
        fill_array(array, SIZE,SIZE);
        display_array("array", array,100);

        results = (long*) malloc( BLOCKS * sizeof(long) );

        hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
        hipMalloc( (void**) &d_r, BLOCKS * sizeof(long) );

        hipMemcpy(d_a, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

        printf("Starting...\n");
        ms = 0;
        for (i = 1; i <= N; i++) {
                start_timer();

                sum<<<BLOCKS, THREADS>>> (d_a, d_r);

                ms += stop_timer();
        }

        hipMemcpy(results, d_r, BLOCKS * sizeof(long), hipMemcpyDeviceToHost);

        long acum = 0;
        for (i = 0; i < BLOCKS; i++) {
                acum += results[i];
        }

        printf("sum = %li\n", acum);
        printf("avg time = %.5lf\n", (ms / N));

        hipFree(d_r);
        hipFree(d_a);

        free(array);
        free(results);
        return 0;
}

// implement your code
