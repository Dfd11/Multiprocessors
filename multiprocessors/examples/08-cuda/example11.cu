#include "hip/hip_runtime.h"
// =================================================================
//
// File: example11.cpp
// Author(s):
// A01705291 Ramses Aguila Flores 
// A01209414 David Flores Diaz
// speedup = 7.68680 ms / 0.00310 ms = 247961.29 %
// Description: This file implements the code that transforms a
//				grayscale image. Uses OpenCV, to compile:
//				g++ example11.cpp `pkg-config --cflags --libs opencv4`
//
//				The time this implementation takes will be used as the
//				basis to calculate the improvement obtained with
//				parallel technologies.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include "utils.h"
typedef unsigned char uchar;

__global__ void gray(uchar *src, uchar *dest, int size) { 

	int tid = ((blockIdx.x * blockDim.x) + threadIdx.x) * 3;
	while (tid < size){
		int val = 0;
		val += src[tid + 0];
		val += src[tid + 1];
		val += src[tid + 2];

		val /= 3;

		dest[tid + 0] = val;
		dest[tid + 1] = val;
		dest[tid + 2] = val;

		tid += blockDim.x * gridDim.x * 3;
	}
}

int main(int argc, char* argv[]) {
	int i;
	double acum; 
 	uchar *dev_src, *dev_dest;
		
	cv::Mat  src = cv::imread(argv[1],cv::IMREAD_COLOR);
	cv::Mat dest = cv::Mat(src.rows, src.cols, CV_8UC3);

  	long size = src.rows * src.cols * 3;

	hipMalloc((void**) &dev_src, size * sizeof(uchar));
	hipMalloc((void**) &dev_dest, size * sizeof(uchar));
	
	hipMemcpy(dev_src, src.data, size * sizeof(uchar), hipMemcpyHostToDevice);
	
	acum = 0;
	printf("Starting...\n");
	for (i = 0; i < N; i++) {
		start_timer();

		gray<<<src.rows, src.cols>>>(dev_src, dev_dest, size);

		acum += stop_timer();
	}
	
	hipMemcpy(dest.data, dev_dest, size*sizeof(uchar), hipMemcpyDeviceToHost);
	
	hipFree(dev_dest);
	hipFree(dev_src);
	
	printf("avg time = %.5lf ms\n", (acum / N));

	cv::imwrite("gray.bmp", dest);

	return 0;
}
