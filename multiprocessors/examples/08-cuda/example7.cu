#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Author(s):
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
// A01705291 Ramses Aguila Flores 
// A01209414 David Flores Diaz
// speedup = 140.09380 ms / 0.00220 ms = 6367900 %
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000 //1e6
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__device__ bool prime(long n){
	long lim;
	if(n<2){
		return false;
	}else{
		lim = sqrt((float)n);
		for (long i=2; i<=lim;i++){
			if(n % i == 0){
				return false;
			}
		}
		return true;
	}
}

__global__ void sum(int *array, long *result) {
        __shared__ long cache[THREADS];

        long tid = threadIdx.x + (blockIdx.x * blockDim.x);
        long cacheIndex = threadIdx.x;

        long acum = 0;
        while (tid < SIZE) {
		if (prime(array[tid])){
                	acum += array[tid];
		}
	        tid += blockDim.x * gridDim.x;
        }
        cache[cacheIndex] = acum;

        __syncthreads();

        long i = blockDim.x / 2;
        while (i > 0) {
                if (cacheIndex < i) {
                        cache[cacheIndex] += cache[cacheIndex + i];
                }
                __syncthreads();
                i /= 2;
        }

        if (cacheIndex == 0) {
                result[blockIdx.x] = cache[cacheIndex];
        }
}

int main(int argc, char* argv[]) {
        int i, *array, *d_a;
        long *results, *d_r;
        double ms;

        array = (int*) malloc( SIZE * sizeof(int) );
        fill_array(array, SIZE,SIZE);
        display_array("array", array,100);

        results = (long*) malloc( BLOCKS * sizeof(long) );

        hipMalloc( (void**) &d_a, SIZE * sizeof(long) );
        hipMalloc( (void**) &d_r, BLOCKS * sizeof(long) );

        hipMemcpy(d_a, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

        printf("Starting...\n");
        ms = 0;
        for (i = 1; i <= 1; i++) {
                start_timer();

                sum<<<BLOCKS,THREADS>>> (d_a, d_r);

                ms += stop_timer();
        }

        hipMemcpy(results, d_r, BLOCKS * sizeof(long), hipMemcpyDeviceToHost);

        long acum = 0;
        for (i = 0; i < BLOCKS; i++) {
                acum += results[i];
        }

        printf("sum = %li\n", acum);
        printf("avg time = %.5lf\n", (ms / N));

        hipFree(d_r);
        hipFree(d_a);

        free(array);
        free(results);
        return 0;
}
