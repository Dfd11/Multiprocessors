#include "hip/hip_runtime.h"
// =================================================================
//
// File: example8.cu
// Author(s):
// Description: This file contains the code that implements the
//				enumeration sort algorithm using CUDA.
// A01705291 Ramses Aguila Flores 
// A01209414 David Flores Diaz
// speedup = 418.82030 ms / 0.00170 ms = 24635470.58 %
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 10000
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void sum(int *array, int *result) {

        int tid = threadIdx.x + (blockIdx.x * blockDim.x);


        while (tid < SIZE) {
		////////////////
		int k =0;
		for (int i=0; i < SIZE ; i++){
			if(array[tid] > array[i]){
				k++;
			}else if(array[tid] == array[i] && tid < i){
				k++;
			}
		}
		result[k] = array[tid];
		/////////////////
                tid += blockDim.x * gridDim.x;
        }

}

int main(int argc, char* argv[]) {
        int i, *array, *d_a;
        int *results, *d_r;
        double ms;

        array = (int*) malloc( SIZE * sizeof(int) );
        random_array(array, SIZE);
        display_array("array", array,100);

        results = (int*) malloc( SIZE * sizeof(int) );

        hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
        hipMalloc( (void**) &d_r, SIZE * sizeof(int) );

        hipMemcpy(d_a, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

        printf("Starting...\n");
        ms = 0;
        for (i = 1; i <= N; i++) {
                start_timer();

                sum<<<BLOCKS, THREADS>>> (d_a, d_r);

                ms += stop_timer();
        }

        hipMemcpy(results, d_r, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	display_array("Result",results,100);
	printf("avg time = %.5lf\n", (ms / N));

        hipFree(d_r);
        hipFree(d_a);

        free(array);
        free(results);
        return 0;
}
